#include "hip/hip_runtime.h"
#ifndef MBUSU_KERNEL_NOHALO
#define MBUSU_KERNEL_NOHALO

#include "mbusu_dhpccpp.hpp"
#include "mbusu_kernel_basic.cu"
#ifdef CUDA_VERSION_TILED_NO_HALO

// ----------------------------------------------------------------------------
// CUDA KERNEL ROUTINES
// ----------------------------------------------------------------------------

__global__
void compute_flows_kernel( double *d__substates__, bool substates_swap ____SLICE_LIMIT_PARAMS____ )
{
  const int i = blockIdx.y*blockDim.y + threadIdx.y;
  const int j = blockIdx.x*blockDim.x + threadIdx.x;
  const int k = blockIdx.z*blockDim.z + threadIdx.z;

  ____SLICE_LIMIT_CHECK____

  if ( i >= ROWS || j >= COLS || k >= SLICES )
    return;
  
  Substates d__Q;
  extern __shared__ double s__Q_h[];

  d__Q.__substates__ = d__substates__;
  syncSubstatesPtrs( d__Q, substates_swap );

  SET3D( s__Q_h, blockDim.y, blockDim.x, threadIdx.y, threadIdx.x, threadIdx.z, GET3D(d__Q.h, ROWS, COLS, i, j, k) );
  __syncthreads();

  //
  // Apply the flow computation kernel to the whole domain
  //
  compute_flows( i, j, k, threadIdx.y, threadIdx.x, threadIdx.z, d__Q, s__Q_h,
                 blockDim.y, blockDim.x, blockDim.z );
}


__global__
void mass_balance_kernel( double *d__substates__, Parameters *d__P, bool substates_swap ____SLICE_LIMIT_PARAMS____ )
{
  const int i = blockIdx.y*blockDim.y + threadIdx.y + 1;
  const int j = blockIdx.x*blockDim.x + threadIdx.x + 1;
  const int k = blockIdx.z*blockDim.z + threadIdx.z;

  ____SLICE_LIMIT_CHECK____
  
  if ( i >= (ROWS-1) || j >= (COLS-1) || k >= SLICES )
    return;
  
  Substates d__Q;
  extern __shared__ double s__Q_k[];

  d__Q.__substates__ = d__substates__;
  syncSubstatesPtrs( d__Q, substates_swap );

  #define BLOCK_I (threadIdx.y)
  #define BLOCK_J (threadIdx.x)
  #define BLOCK_K (threadIdx.z)
  #define LAST_BLOCK_I (blockDim.y-1)
  #define LAST_BLOCK_J (blockDim.x-1)
  #define LAST_BLOCK_K (blockDim.z-1)
  #define LAST_I (ROWS-1)
  #define LAST_J (COLS-1)
  #define LAST_K (SLICES-1)
  
  SET3D( s__Q_k, blockDim.y, blockDim.x, threadIdx.y, threadIdx.x, threadIdx.z, GET3D(d__Q.k, ROWS, COLS, i, j, k) );  
  __syncthreads();

  //
  // Apply the mass balance kernel to the domain bounded by mb_bounds 
  //
  mass_balance( i, j, k, threadIdx.y, threadIdx.x, threadIdx.z, d__Q, s__Q_k,
                blockDim.y, blockDim.x, blockDim.z, *d__P );
}

#endif
#endif